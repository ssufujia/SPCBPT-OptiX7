#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
#include <optix.h>

#include <cuda/LocalGeometry.h>
#include <cuda/LocalShading.h>
#include <cuda/helpers.h>
#include <cuda/random.h>
#include <sutil/vec_math.h>
#include "BDPTVertex.h"
#include "cuProg.h"
#include "pathControl.h"
#include "rmis.h"
//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------
 
__device__ inline float4 ToneMap_exposure(const float4& c, float exposure)
{
    float3 ldr = make_float3(1.0) - make_float3(exp(-c.x * exposure), exp(-c.y * exposure), exp(-c.z * exposure));
    return make_float4(ldr.x, ldr.y, ldr.z, 1.0f);
}
__device__ inline float4 ToneMap(const float4& c, float limit)
{
    //return ToneMap_exposure(c,limit);

    float luminance = 0.3f * c.x + 0.6f * c.y + 0.1f * c.z;

    float4 col = c * 1.0f / (1.0f + 1 * luminance / limit);
    return make_float4(col.x, col.y, col.z, 1.0f);
}

__device__ inline float color2luminance(const float3& c)
{
    return 0.3f * c.x + 0.6f * c.y + 0.1f * c.z;
}
__device__ inline float4 LinearToSrgb(const float4& c)
{
    const float kInvGamma = 1.0f / 2.2f;
    return make_float4(powf(c.x, kInvGamma), powf(c.y, kInvGamma), powf(c.z, kInvGamma), c.w);
}

/* Õâ¸öº¯ÊýÓ¦¸ÃÊÇ PT */
extern "C" __global__ void __raygen__pinhole()
{  
    const uint3  launch_idx = optixGetLaunchIndex();
    const uint3  launch_dims = optixGetLaunchDimensions();
    const float3 eye = Tracer::params.eye;
    const float3 U = Tracer::params.U;
    const float3 V = Tracer::params.V;
    const float3 W = Tracer::params.W;
    const int    subframe_index = Tracer::params.subframe_index;
     
    float3 normalizeV = normalize(V);

    /* Generate camera ray */
    unsigned int seed = tea<4>( launch_idx.y * launch_dims.x + launch_idx.x, subframe_index );

    /* The center of each pixel is at fraction(0.5, 0.5) */
    const float2 subpixel_jitter =
        subframe_index == 0 ? make_float2( 0.5f, 0.5f ) : make_float2( rnd( seed ), rnd( seed ) );

    const float2 d =
        2.0f * make_float2( ( static_cast<float>( launch_idx.x ) + subpixel_jitter.x ) / static_cast<float>( launch_dims.x ),
                           ( static_cast<float>( launch_idx.y ) + subpixel_jitter.y ) / static_cast<float>( launch_dims.y ) ) - 1.0f;

    float3 ray_direction = normalize( d.x * U + d.y * V + W );
    float3 ray_origin    = eye;

    /* Trace camera ray */
    /* ÕâÀï payload ÔÚ´´½¨Ê±±»²¿·Ö³õÊ¼»¯ÁË */
    Tracer::PayloadRadiance payload; 
    payload.seed          = seed; 
    payload.origin        = eye;
    payload.ray_direction = ray_direction;
    payload.currentResult = make_float3(0);
    while (true)
    {
        ray_direction = payload.ray_direction;
        ray_origin = payload.origin; 
        Tracer::traceRadiance(Tracer::params.handle, ray_origin, ray_direction,
            SCENE_EPSILON,  // tmin
            1e16f,  // tmax
            &payload
        );

        /* ÕâÀï¿É¼ûÐÔ²âÊÔËÆºõÊÇ¶Ô NEE ×öµÄ */
        if (float3weight(payload.currentResult)> 0.0)
        {
            const float  L_dist = length(payload.vis_pos_A- payload.vis_pos_B);
            const float3 L = (payload.vis_pos_B - payload.vis_pos_A) / L_dist;
            if (Tracer::visibilityTest(Tracer::params.handle, payload.vis_pos_A, payload.vis_pos_B))
                payload.result += payload.currentResult; 
            payload.currentResult = make_float3(0);
        }
        if (payload.done || payload.depth > 30)
            break;
        payload.depth += 1;
    }

    /* Update results */
    const unsigned int image_index = launch_idx.y * launch_dims.x + launch_idx.x;
    float3             accum_color = payload.result;

    if( subframe_index > 0 )
    {
        const float  a                = 1.0f / static_cast<float>( subframe_index + 1 );
        const float3 accum_color_prev = make_float3( Tracer::params.accum_buffer[image_index] );
        accum_color                   = lerp( accum_color_prev, accum_color, a );
    }
    //if (subframe_index > 1000)return;
    Tracer::params.accum_buffer[image_index] = make_float4( accum_color, 1.0f );

    float4 val = ToneMap(make_float4(accum_color, 0.0), 1.5);
    Tracer::params.frame_buffer[image_index] = make_color( make_float3(val) );
} 

RT_FUNCTION void init_lightSubPath_from_lightSample(Tracer::lightSample& light_sample, BDPTPath& p)
{
    p.clear();
    p.push();
    BDPTVertex& v = p.currentVertex();
     
    if (light_sample.bindLight->type == Light::Type::QUAD)
    {
        p.nextVertex().singlePdf = light_sample.dir_pdf;
    }
    else if (light_sample.bindLight->type == Light::Type::ENV)
    {
        p.nextVertex().singlePdf = light_sample.dir_pos_pdf;
    }

    init_vertex_from_lightSample(light_sample, v);
     //ÆäËû¹âÔ´µÄ×´¿ö´ý²¹³ä
}


__device__ float3 direction_connect_ZGCBPT(const BDPTVertex& a, const BDPTVertex& b)
{
    float3 L = make_float3(0.0f);
    float3 connectDir = -b.normal; 
    if (dot(a.normal, connectDir) > 0.0)
    {
        MaterialData::Pbr mat_a = Tracer::params.materials[a.materialId];
        mat_a.base_color = make_float4(a.color, 1.0);
        float3 f = Tracer::Eval(mat_a, a.normal, normalize(a.lastPosition - a.position), connectDir)
            * dot(a.normal, connectDir);
        L = a.flux / a.pdf * f * b.flux / b.pdf * rmis::connection_direction_lightSource(a, b);
    }
    if (ISINVALIDVALUE(L))
    {
        return make_float3(0.0f);
    }
    return L;

}
__device__  float3 connectVertex_SPCBPT(const BDPTVertex& a, const BDPTVertex& b)
{
    if (b.is_DIRECTION())
    {
        return direction_connect_ZGCBPT(a, b);
    }
    float3 connectVec = a.position - b.position;
    float3 connectDir = normalize(connectVec);
    float G = abs(dot(a.normal, connectDir)) * abs(dot(b.normal, connectDir)) / dot(connectVec, connectVec);
    float3 LA = a.lastPosition - a.position;
    float3 LA_DIR = normalize(LA);
    float3 LB = b.lastPosition - b.position;
    float3 LB_DIR = normalize(LB); 

    float3 fa, fb;
    float3 ADcolor;
    MaterialData::Pbr mat_a = Tracer::params.materials[a.materialId];
    mat_a.base_color = make_float4(a.color, 1.0);  
    fa = Tracer::Eval(mat_a, a.normal, LA_DIR, -connectDir) / (mat_a.brdf ? abs(dot(a.normal, connectDir)) : 1.0f);

    MaterialData::Pbr mat_b;
    if (!b.isOrigin)
    {
        mat_b = Tracer::params.materials[b.materialId];
        mat_b.base_color = make_float4(b.color,1.0);
        fb = Tracer::Eval(mat_b, b.normal, connectDir, LB_DIR) / (mat_b.brdf ? abs(dot(b.normal, connectDir)) : 1.0f);
    }
    else
    {
        if (dot(b.normal, -connectDir) > 0.0f)
        {
            fb = make_float3(0.0f);
        }
        else
        {
            fb = make_float3(1.0f);
        }
    }
    float3 temp_vec = a.flux / a.pdf;
    //printf("connect info %f %f %f\n", temp_vec.x, temp_vec.y, temp_vec.z);
    float3 contri = a.flux * b.flux * fa * fb * G;
    float pdf = a.pdf * b.pdf; 
    float3 ans = contri / pdf *(b.depth == 0 ? rmis::connection_lightSource(a, b) : rmis::general_connection(a, b));


    if (ISINVALIDVALUE(ans))
    {
        return make_float3(0.0f);
    }
    return  ans;
}

RT_FUNCTION float3 lightStraghtHit(BDPTVertex& a)
{
    float3 contri = a.flux;
    float pdf = a.pdf;
    float inver_weight = a.RMIS_pointer;

    float3 ans = contri / pdf / inver_weight;
    if (ISINVALIDVALUE(ans))
    {
        return make_float3(0.0f);
    }
    return  ans;
}

extern "C" __global__ void __raygen__SPCBPT()
{
    const uint3  launch_idx = optixGetLaunchIndex();
    const uint3  launch_dims = optixGetLaunchDimensions();
    const float3 eye = Tracer::params.eye;
    const float3 U = Tracer::params.U;
    const float3 V = Tracer::params.V;
    const float3 W = Tracer::params.W;
    const int    subframe_index = Tracer::params.subframe_index;

    float3 normalizeV = normalize(V); 
    // Generate camera ray
    //
    unsigned int seed = tea<4>(launch_idx.y * launch_dims.x + launch_idx.x, subframe_index);

    // The center of each pixel is at fraction (0.5,0.5)
    const float2 subpixel_jitter =
        subframe_index == 0 ? make_float2(0.5f, 0.5f) : make_float2(rnd(seed), rnd(seed));

    const float2 d =
        2.0f
        * make_float2((static_cast<float>(launch_idx.x) + subpixel_jitter.x) / static_cast<float>(launch_dims.x),
            (static_cast<float>(launch_idx.y) + subpixel_jitter.y) / static_cast<float>(launch_dims.y))
        - 1.0f;
    float3 ray_direction = normalize(d.x * U + d.y * V + W);
    float3 ray_origin = eye;
    float3 result = make_float3(0);

    Tracer::PayloadBDPTVertex payload;
    payload.clear();
    payload.seed = seed; 
    payload.ray_direction = ray_direction;
    payload.origin = ray_origin;
    
    init_EyeSubpath(payload.path, ray_origin, ray_direction);

       
    unsigned first_hit_id;
    while (true)
    {
        ray_direction = payload.ray_direction;
        ray_origin = payload.origin;
        if (payload.done || payload.depth > 50)
        {
            break;
        }
        int begin_depth = payload.path.size;
        Tracer::traceEyeSubPath(Tracer::params.handle, ray_origin, ray_direction,
            SCENE_EPSILON,  // tmin
            1e16f,  // tmax
            &payload);
        if (payload.path.size == begin_depth)
        {
            break;
        }
        payload.depth += 1;


        //if (payload.path.size == 2)
        //{
        //    BDPTVertex v = payload.path.currentVertex();
        //    labelUnit lu(v.position, v.normal, normalize(v.lastPosition - v.position), false);
        //    first_hit_id = lu.getLabel();
        //}  
        if (payload.path.hit_lightSource())
        {
            float3 res = lightStraghtHit(payload.path.currentVertex());
            result += res;
            break;
        }
        BDPTVertex& eye_subpath = payload.path.currentVertex();
        for (int it = 0; it < CONNECTION_N; it++)
        {

            int light_id = 0;
            float pmf_firstStage = 1;
            if (Tracer::params.subspace_info.light_tree)
            {
                light_id =
                    reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->sampleFirstStage(eye_subpath.subspaceId, payload.seed, pmf_firstStage);
            }
            if (Tracer::params.sampler.subspace[light_id].size == 0)
            {
                continue;
            }
            float pmf_secondStage;
            const BDPTVertex& light_subpath =
                reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->sampleSecondStage(light_id, payload.seed, pmf_secondStage);

            if ((Tracer::visibilityTest(Tracer::params.handle, eye_subpath, light_subpath)))
            { 
                //printf("debug info %f\n", float3weight(tmp_float3));
                float pmf = Tracer::params.sampler.path_count * pmf_secondStage * pmf_firstStage;
                float3 res = connectVertex_SPCBPT(eye_subpath, light_subpath) / pmf;
                if (!ISINVALIDVALUE(res))
                {
                    result += res / CONNECTION_N;
                }
            }
     
        } 
        //printf("%d size error depth%d\n", Tracer::params.lights.count, payload.path.size);
    } 
    
    //env map
    result += payload.result;

    //
    // Update results 
    ////  
    //result = make_float3(rnd(first_hit_id), rnd(first_hit_id), rnd(first_hit_id));  
    const unsigned int image_index = launch_idx.y * launch_dims.x + launch_idx.x;
    float3             accum_color = result;

    if (subframe_index > 0)
    {
        const float  a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(Tracer::params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    } 
    Tracer::params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
     
    float4 val = ToneMap(make_float4(accum_color, 0.0), 1.5);
    Tracer::params.frame_buffer[image_index] = make_color(make_float3(val));  
}

RT_FUNCTION float3 eval_path(const BDPTVertex* path, int path_size, int strategy_id)
{
    //return Tracer::contriCompute(path,path_size);
    float pdf = Tracer::pdfCompute(path, path_size, strategy_id);
    float3 contri = Tracer::contriCompute(path, path_size);
    float3 ans = contri / pdf;

    //mis computation
    {
        float MIS_weight_not_normalize = Tracer::MISWeight_SPCBPT(path, path_size, strategy_id);
        float MIS_weight_dominator = 0.0;
        for (int i = 2; i <= path_size; i++)
        {
            MIS_weight_dominator += Tracer::MISWeight_SPCBPT(path, path_size, i);
        }

        ans = ans * (MIS_weight_not_normalize / MIS_weight_dominator);
    } 
    if (ISINVALIDVALUE(ans))
    {
        return make_float3(0.0f);
    }
    return ans;
}
extern "C" __global__ void __raygen__SPCBPT_no_rmis()
{
    const uint3  launch_idx = optixGetLaunchIndex();
    const uint3  launch_dims = optixGetLaunchDimensions();
    const float3 eye = Tracer::params.eye;
    const float3 U = Tracer::params.U;
    const float3 V = Tracer::params.V;
    const float3 W = Tracer::params.W;
    const int    subframe_index = Tracer::params.subframe_index;

    float3 normalizeV = normalize(V);
    // Generate camera ray
    //
    unsigned int seed = tea<4>(launch_idx.y * launch_dims.x + launch_idx.x, subframe_index);

    // The center of each pixel is at fraction (0.5,0.5)
    const float2 subpixel_jitter =
        subframe_index == 0 ? make_float2(0.5f, 0.5f) : make_float2(rnd(seed), rnd(seed));

    const float2 d =
        2.0f
        * make_float2((static_cast<float>(launch_idx.x) + subpixel_jitter.x) / static_cast<float>(launch_dims.x),
            (static_cast<float>(launch_idx.y) + subpixel_jitter.y) / static_cast<float>(launch_dims.y))
        - 1.0f;
    float3 ray_direction = normalize(d.x * U + d.y * V + W);
    float3 ray_origin = eye;
    float3 result = make_float3(0);

    Tracer::PayloadBDPTVertex payload;
    payload.clear();
    payload.seed = seed;
    payload.ray_direction = ray_direction;
    payload.origin = ray_origin;
    init_EyeSubpath(payload.path, ray_origin, ray_direction);


#define MAX_PATH_LENGTH_FOR_MIS 8
    BDPTVertex pathBuffer[MAX_PATH_LENGTH_FOR_MIS +4];
    int buffer_size = 0;
    pathBuffer[buffer_size] = payload.path.currentVertex(); buffer_size++; 

    unsigned first_hit_id;
    while (true)
    {
        ray_direction = payload.ray_direction;
        ray_origin = payload.origin;
        if (payload.done || payload.depth > 50)
            break;
        int begin_depth = payload.path.size;
        Tracer::traceEyeSubPath(Tracer::params.handle, ray_origin, ray_direction,
            SCENE_EPSILON,  // tmin
            1e16f,  // tmax
            &payload);
        if (payload.path.size == begin_depth)
        {
            break;
        } 
        payload.depth += 1;


        pathBuffer[buffer_size] = payload.path.currentVertex(); buffer_size++;
        if (payload.path.hit_lightSource())
        {  
            if (RMIS_FLAG)
            { 
                float3 res = lightStraghtHit(payload.path.currentVertex());
                result += res;
            }
            else
            {

                float3 res = make_float3(0.0);
                Tracer::lightSample light_sample;
                light_sample.ReverseSample(Tracer::params.lights[payload.path.currentVertex().materialId], payload.path.currentVertex().uv);

                BDPTVertex light_vertex;
                init_vertex_from_lightSample(light_sample, light_vertex);
                pathBuffer[buffer_size - 1] = light_vertex;

                res = eval_path(pathBuffer, buffer_size, buffer_size);
                if (buffer_size > MAX_PATH_LENGTH_FOR_MIS)
                    res *= 0;
                result += res;
            }
            break;
        }
        if (buffer_size >= MAX_PATH_LENGTH_FOR_MIS + 4) 
            break;

        BDPTVertex& eye_subpath = payload.path.currentVertex();
        for (int it = 0; it < CONNECTION_N; it++)
        {

            int light_id = 0;
            float pmf_firstStage = 1;
            if (Tracer::params.subspace_info.light_tree)
            {
                light_id =
                    reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->sampleFirstStage(eye_subpath.subspaceId, payload.seed, pmf_firstStage);
            }
            if (Tracer::params.sampler.subspace[light_id].size == 0)
            {
                continue;
            }
            float pmf_secondStage;
            const BDPTVertex& light_subpath =
                reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->sampleSecondStage(light_id, payload.seed, pmf_secondStage);

            if ((buffer_size + light_subpath.depth + 1 <= MAX_PATH_LENGTH_FOR_MIS ) &&
                (Tracer::visibilityTest(Tracer::params.handle, eye_subpath.position, light_subpath.position)))
            { 
                float pmf = Tracer::params.sampler.path_count * pmf_secondStage * pmf_firstStage;
                

                float3 res;
                if (RMIS_FLAG)
                {
                    res = connectVertex_SPCBPT(eye_subpath, light_subpath) / pmf;
                }
                else
                {

                    int origin_buffer_size = buffer_size;
                    const BDPTVertex* light_ptr = &light_subpath;
                    while (true)
                    {
                        pathBuffer[buffer_size] = *light_ptr; buffer_size++;
                        if (light_ptr->depth == 0)break;
                        light_ptr--;
                    }
                    res = eval_path(pathBuffer, buffer_size, origin_buffer_size) / pmf;

                    buffer_size = origin_buffer_size;
                }
                
                 
                if (!ISINVALIDVALUE(res))
                {
                    result += res / CONNECTION_N;
                }
            }
        }
        //printf("%d size error depth%d\n", Tracer::params.lights.count, payload.path.size);
    }
    //
    // Update results 
    ////  
    //result = make_float3(rnd(first_hit_id), rnd(first_hit_id), rnd(first_hit_id));  
    const unsigned int image_index = launch_idx.y * launch_dims.x + launch_idx.x;
    float3             accum_color = result;

    if (subframe_index > 0)
    {
        const float  a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(Tracer::params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);

        if (accum_color.x < 0 || accum_color.y < 0 || accum_color.z < 0)
        {
            accum_color = accum_color_prev;
        }
    }

    if (accum_color.x < 0 || accum_color.y < 0 || accum_color.z < 0)
    {
        accum_color = make_float3(0);
    }
    Tracer::params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
   
    float4 val = ToneMap(make_float4(accum_color, 0.0), 1.5);
    Tracer::params.frame_buffer[image_index] = make_color(make_float3(val));
}

extern "C" __global__ void __raygen__shift_combine()
{
    const uint3  launch_idx = optixGetLaunchIndex();
    const uint3  launch_dims = optixGetLaunchDimensions();
    const float3 eye = Tracer::params.eye;
    const float3 U = Tracer::params.U;
    const float3 V = Tracer::params.V;
    const float3 W = Tracer::params.W;
    const int    subframe_index = Tracer::params.subframe_index;

    float3 normalizeV = normalize(V);
    // Generate camera ray
    unsigned int seed = tea<4>(launch_idx.y * launch_dims.x + launch_idx.x, subframe_index);

    // The center of each pixel is at fraction (0.5,0.5)
    const float2 subpixel_jitter =
        subframe_index == 0 ? make_float2(0.5f, 0.5f) : make_float2(rnd(seed), rnd(seed));

    const float2 d =
        2.0f * make_float2((static_cast<float>(launch_idx.x) + subpixel_jitter.x) / static_cast<float>(launch_dims.x),
            (static_cast<float>(launch_idx.y) + subpixel_jitter.y) / static_cast<float>(launch_dims.y)) - 1.0f;
    float3 ray_direction = normalize(d.x * U + d.y * V + W);
    float3 ray_origin = eye;
    float3 result = make_float3(0);

    Tracer::PayloadBDPTVertex payload;
    payload.clear();
    payload.seed = seed;
    payload.ray_direction = ray_direction;
    payload.origin = ray_origin;
    /* 视子路初始化 */  
    init_EyeSubpath(payload.path, ray_origin, ray_direction);

    BDPTVertex pathBuffer[MAX_PATH_LENGTH_FOR_MIS];
    int buffer_size = 0;
    pathBuffer[buffer_size++] = payload.path.currentVertex();

    unsigned first_hit_id;

    /* 视子路追踪主循环 */
    while (true)
    {
        ray_direction = payload.ray_direction;
        ray_origin = payload.origin;
        if (payload.done || payload.depth > 50)
            break;
        int begin_depth = payload.path.size;
        /* 视子路追踪 */
        Tracer::traceEyeSubPath(
            Tracer::params.handle, ray_origin, ray_direction,
            SCENE_EPSILON,  // tmin
            1e16f,  // tmax
            &payload
        );
        
        /* 没打中 */
        if (payload.path.size == begin_depth)
            break;

        /* 记录一下历史路径 */
        payload.path_record = (payload.path_record) |
            ((long long)Shift::glossy(payload.path.currentVertex()) << payload.depth);
        payload.depth += 1;
        pathBuffer[buffer_size++] = payload.path.currentVertex(); 

        /* 如果打中了光源 */
        float3 res = make_float3(0.0);
        if (payload.path.hit_lightSource())
        {
            if (false&&RMIS_FLAG)
            {
                res = lightStraghtHit(payload.path.currentVertex());
            }
            else
            {
                res = make_float3(0.0);
                Tracer::lightSample light_sample;
                light_sample.ReverseSample(Tracer::params.lights[payload.path.currentVertex().materialId], payload.path.currentVertex().uv);

                BDPTVertex light_vertex;
                init_vertex_from_lightSample(light_sample, light_vertex);
                pathBuffer[buffer_size - 1] = light_vertex;
                res += eval_path(pathBuffer, buffer_size, buffer_size);
            } 
            /* 下 BDPT control 只有光源直击会被保留*/
            if (BDPT_CONTROL)
            {
                if (LE_ENABLE)
                {
                    if (payload.depth != 1)
                        res *= 0;
                }
                else
                    res *= 0;

            }
            result += res;
            break;
        }
        
        if (buffer_size >= MAX_PATH_LENGTH_FOR_MIS) 
            break;

        BDPTVertex& eye_vertex = payload.path.currentVertex();

        /* 视子路和光子路连接 */
        for (int it = 0; it < CONNECTION_N; it++)
        {
            float caustic_connection_prob;
            if (BDPT_CONTROL && S_ONLY)
                caustic_connection_prob = 1;
            else 
            {
                caustic_connection_prob = Tracer::params.subspace_info.caustic_ratio[eye_vertex.subspaceId];
                float b = 1 - caustic_connection_prob;
                caustic_connection_prob *= 10;
                caustic_connection_prob = caustic_connection_prob / (caustic_connection_prob + b);
                // caustic_connection_prob = max(caustic_connection_prob, .5);
                // caustic_connection_prob = .5;
            }

            if (rnd(payload.seed) < caustic_connection_prob)
            {
                float pmf_firstStage = 1;
                float pmf_secondStage;
                float pmf_uniform;
                float guide_ratio = 1 - CONSERVATIVE_RATE;
                guide_ratio = 0;
                const BDPTVertex* light_subpath_p;
                if (rnd(payload.seed) > guide_ratio)
                {
                    if (Tracer::params.sampler.glossy_count == 0)continue;
                    const BDPTVertex& light_subpath =
                        reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->uniformSampleGlossy(payload.seed, pmf_uniform);
                    light_subpath_p = &light_subpath;

                    pmf_firstStage = Tracer::params.subspace_info.CMFCausticGamma[eye_vertex.subspaceId * NUM_SUBSPACE + light_subpath.subspaceId];
                    if (light_subpath.subspaceId != 0)
                        pmf_firstStage -= Tracer::params.subspace_info.CMFCausticGamma[eye_vertex.subspaceId * NUM_SUBSPACE + light_subpath.subspaceId - 1];
                    pmf_secondStage = 1.0 / Tracer::params.sampler.glossy_subspace_num[light_subpath.subspaceId];
                }
                else
                {
                    int light_subspaceId =
                        reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->SampleGlossyFirstStage(eye_vertex.subspaceId, payload.seed, pmf_firstStage);
                    if (Tracer::params.sampler.glossy_subspace_num[light_subspaceId] == 0)continue;
                    const BDPTVertex& light_subpath =
                        reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->sampleSecondStage(light_subspaceId, payload.seed, pmf_secondStage);
                    light_subpath_p = &light_subpath;

                    pmf_uniform = 1.0 / Tracer::params.sampler.glossy_count;
                }
                const BDPTVertex& light_subpath = *light_subpath_p;

                float final_pmf = guide_ratio * (pmf_firstStage * pmf_secondStage) + (1 - guide_ratio) * pmf_uniform; 

                if (
                    /* LSAE，A 代表 any */
                    ((LSAE_ENABLE && light_subpath.depth == 1) ||
                    /* LSDE */
                    (LSDE_ENABLE && light_subpath.depth == 1 && payload.depth == 1 && !payload.path_record) ||
                    /* LDSDE，光子路LDS，视子路DE */
                    (LDSDE_ENABLE && light_subpath.depth == 2 && light_subpath.path_record == 0b10 && payload.depth == 1 &&  !payload.path_record)) &&
                    /* 其他约束条件 */
                    (buffer_size + light_subpath.depth + 1 <= MAX_PATH_LENGTH_FOR_MIS) &&
                    (Tracer::visibilityTest(Tracer::params.handle, eye_vertex.position, light_subpath.position)))
                {
                    // printf("here\n");
                    const BDPTVertex* light_ptr = &light_subpath;
                    float pmf = Tracer::params.sampler.path_count * final_pmf * caustic_connection_prob;

                    if (light_subpath.depth < SHIFT_VALID_SIZE - 1)
                    {
                        BDPTVertex light_sub_new[SHIFT_VALID_SIZE];
                        Shift::PathContainer originPath(const_cast<BDPTVertex*>(&light_subpath), -1, light_subpath.depth + 1);
                        Shift::PathContainer finalPath(light_sub_new, 1);

                        float pdf_retrace;
                        finalPath.setSize(originPath.size());
                        /* 0 号是 glossy 顶点*/
                        finalPath.get(0) = originPath.get(0);

                        /*  LS 光子路 */
                        if (light_subpath.depth == 1) 
                        {
                            BDPTVertex np;
                            MaterialData::Pbr mat = VERTEX_MAT(finalPath.get(0));
                            float3 in_dir = normalize(eye_vertex.position - finalPath.get(0).position);
                            float3 out_dir = Tracer::Sample(mat, finalPath.get(0).normal, in_dir, seed); 

                            bool trace_success = 0;
                            np = Tracer::FastTrace(finalPath.get(0), out_dir, trace_success);
                            /* Ã»´òµ½ */
                            if (trace_success == false) continue;
                            /* Ã»´òµ½¹âÔ´ */
                            if (np.type != BDPTVertex::Type::HIT_LIGHT_SOURCE) continue;

                            Light light = Tracer::params.lights[np.materialId];
                            Tracer::lightSample light_sample;
                            
                           light_sample.ReverseSample(light, np.uv);
                            init_vertex_from_lightSample(light_sample, np);

                            pdf_retrace = Tracer::Pdf(mat, finalPath.get(0).normal, in_dir, out_dir) *
                                 Shift::GeometryTerm(finalPath.get(0), np) / abs(dot(out_dir,finalPath.get(0).normal)); 

                            /* 新的光源顶点 */
                            finalPath.get(1) = np;
                        }
                        /* LDS 光子路，即 S - D - L，path record 为 0b10 */
                        else if (light_subpath.depth == 2 && light_subpath.path_record == 0b10)
                        {
                            /* 0号是glossy顶点，1号是diffuse顶点，2号是光源顶点 */
                            finalPath.get(2) = originPath.get(2);
                            pdf_retrace = 1;

                            BDPTVertex np;
                            MaterialData::Pbr mat = VERTEX_MAT(finalPath.get(0));
                            float3 in_dir = normalize(eye_vertex.position - finalPath.get(0).position);
                            float3 out_dir = Tracer::Sample(mat, finalPath.get(0).normal, in_dir, seed);
                            bool trace_success = 0;
                            np = Tracer::FastTrace(finalPath.get(0), out_dir, trace_success);
                            /* 没追到 */
                            if (trace_success == false) continue;
                            /* 追到了光源 */
                            if (np.type == BDPTVertex::Type::HIT_LIGHT_SOURCE) continue;
                            /* 追到了glossy */
                            if (Shift::glossy(np)) continue;
                            /* 可见性测试 */
                            if (!Tracer::visibilityTest(Tracer::params.handle, np, finalPath.get(2))) continue;

                            finalPath.get(1) = np;

                            pdf_retrace = 1;//Tracer::Pdf(mat, finalPath.get(0).normal, in_dir, out_dir) *
                                //Shift::GeometryTerm(finalPath.get(0), np) / abs(dot(out_dir, finalPath.get(0).normal)); ;
                        }


                        for (int i = 0; i < finalPath.size(); i++)
                            pathBuffer[buffer_size + i] = finalPath.get(i);
                        
                        float pdf = eye_vertex.pdf  * pdf_retrace;

                        float3 contri = Tracer::contriCompute(pathBuffer, buffer_size + finalPath.size()); 
                        //printf("contri:  %f %f %f\n", contri.x, contri.y, contri.z);
                        //printf("pdf: %f", pdf);
                        float3 res = (contri / pdf / pmf) * light_subpath.inverPdfEst;                        

                        if (!ISINVALIDVALUE(res))
                            result += res / CONNECTION_N;
                    }
                    else
                    {
                        continue;
                        int origin_buffer_size = buffer_size;
                        while (true)
                        {
                            pathBuffer[buffer_size] = *light_ptr; buffer_size++;
                            if (light_ptr->depth == 0)break;
                            light_ptr--;
                        }
                        int n_buffer_size = buffer_size;
                        buffer_size = origin_buffer_size;
                        float pdf = Tracer::pdfCompute(pathBuffer, n_buffer_size, origin_buffer_size);
                        float3 contri = Tracer::contriCompute(pathBuffer, n_buffer_size);
                        
                        float3 res = contri / pdf / pmf;

                        if (!ISINVALIDVALUE(res))
                            result += res / CONNECTION_N;
                    }
                }
            }

            /* ·Ç½¹É¢Â·¾¶ */
            else
            {
                if (S_ONLY)
                    continue;

                int light_id = 0;
                float pmf_firstStage = 1;
                if (Tracer::params.subspace_info.light_tree)
                {
                    light_id =
                        reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->sampleFirstStage(eye_vertex.subspaceId, payload.seed, pmf_firstStage);
                }
                if (Tracer::params.sampler.subspace[light_id].size == 0)
                {
                    continue;
                }
                float pmf_secondStage;
                const BDPTVertex& light_subpath =
                    reinterpret_cast<Tracer::SubspaceSampler_device*>(&Tracer::params.sampler)->sampleSecondStage(light_id, payload.seed, pmf_secondStage);
                if (Shift::glossy(light_subpath))continue;
                if ((buffer_size + light_subpath.depth + 1 <= MAX_PATH_LENGTH_FOR_MIS) &&
                    (Tracer::visibilityTest(Tracer::params.handle, eye_vertex.position, light_subpath.position)))
                {
                    float pmf = Tracer::params.sampler.path_count * pmf_secondStage * pmf_firstStage * (1 - caustic_connection_prob);

                    int origin_buffer_size = buffer_size;
                    float3 res;
                    if (RMIS_FLAG)
                    {
                        res = connectVertex_SPCBPT(eye_vertex, light_subpath) / pmf;
                    }
                    else
                    {
                        const BDPTVertex* light_ptr = &light_subpath;
                        while (true)
                        {
                            pathBuffer[buffer_size] = *light_ptr; buffer_size++;
                            if (light_ptr->depth == 0)break;
                            light_ptr--;
                        }
                        res = eval_path(pathBuffer, buffer_size, origin_buffer_size) / pmf;
                    }

                    // float3 res = eval_path(pathBuffer, buffer_size, origin_buffer_size) / pmf;
                    // if (Shift::IsCausticPath(pathBuffer, buffer_size)) res *= 0;
                    buffer_size = origin_buffer_size;

                    if (!ISINVALIDVALUE(res))
                    {
                        result += res / CONNECTION_N;
                    }
                }
            }

        }
    }
    //
    // Update results 
    ////  
    //result = make_float3(rnd(first_hit_id), rnd(first_hit_id), rnd(first_hit_id));  
    const unsigned int image_index = launch_idx.y * launch_dims.x + launch_idx.x;
    float3             accum_color = result;

    if (subframe_index > 0)
    {
        const float  a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(Tracer::params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);

        if (accum_color.x < 0 || accum_color.y < 0 || accum_color.z < 0)
        {
            accum_color = accum_color_prev;
        }
    }

    if (accum_color.x < 0 || accum_color.y < 0 || accum_color.z < 0)
    {
        accum_color = make_float3(0);
    }

    //if (subframe_index > 2000)return;
    Tracer::params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);

    float4 val = ToneMap(make_float4(accum_color, 0.0), 1.5);
    Tracer::params.frame_buffer[image_index] = make_color(make_float3(val));
}

#define CheckLightBufferState if(!(lightVertexCount<lt_params.core_padding)) break; 

RT_FUNCTION void pushVertexToLVC(BDPTVertex& v, unsigned int& putId, int bufferBias)
{
    const LightTraceParams& lt_params = Tracer::params.lt;
    lt_params.ans[putId + bufferBias] = v;
    lt_params.validState[putId + bufferBias] = true;
    putId++;
}
extern "C" __global__ void __raygen__lightTrace()
{
    const uint3  launch_idx = optixGetLaunchIndex();
    const uint3  launch_dims = optixGetLaunchDimensions();
    const int    subframe_index = Tracer::params.lt.launch_frame;
    unsigned int seed = tea<4>(launch_idx.y * launch_dims.x + launch_idx.x, subframe_index);

    Tracer::PayloadBDPTVertex payload;
    payload.seed = seed;

    const LightTraceParams& lt_params = Tracer::params.lt;
    int launch_index = launch_idx.x;
    unsigned int bufferBias = lt_params.core_padding * launch_index;
    unsigned int lightVertexCount = 0;
    unsigned int lightPathCount = 0;
     
    while (true)
    {
        payload.clear();
        int light_id = clamp(static_cast<int>(floorf(rnd(seed) * Tracer::params.lights.count)), 
                                int(0), int(Tracer::params.lights.count - 1));
        const Light& light = Tracer::params.lights[light_id];
        Tracer::lightSample light_sample; 
        light_sample(light, seed); 

        light_sample.traceMode(seed);
        float3 ray_direction = light_sample.trace_direction();
        float3 ray_origin = light_sample.position; 
        init_lightSubPath_from_lightSample(light_sample, payload.path);
        /* lightVertexCount 在经过这个函数后会加 1 */
        pushVertexToLVC(payload.path.currentVertex(), lightVertexCount, bufferBias); 
        CheckLightBufferState;

        /* 光子路追踪 */
        while (true)
        {
            int begin_depth = payload.path.size;
            Tracer::traceLightSubPath(
                Tracer::params.handle, 
                ray_origin, 
                ray_direction,
                SCENE_EPSILON,  // tmin
                1e16f,  // tmax
                &payload
            );
            /* 如果打中了面片 */
            if (payload.path.size > begin_depth) 
            {
                BDPTVertex& curVertex = payload.path.currentVertex();
                /* 将历史信息记录进 pathrecord */
                payload.path_record = (payload.path_record) |
                    ((long long)Shift::glossy(curVertex) << payload.depth);
                curVertex.path_record = payload.path_record;

                /* L -> S 光子路 */
                if (curVertex.depth == 1 && curVertex.path_record)
                {
                    BDPTVertex v[2];
                    /* v[1] 是光源顶点 */
                    v[1] = payload.path.lastVertex();
                    v[0] = curVertex;

                    Shift::PathContainer path(v, 1, 2);
                    float pdf_inverse = Shift::inverPdfEstimate(path, payload.seed);
                    curVertex.inverPdfEst = pdf_inverse;
                } 

                /* L -> D -> S 光子路，即 S - D - L， path_record 为 0b10 */
                else if (curVertex.depth == 2 && curVertex.path_record == 0b10)
                {
                    // TBD
                    curVertex.inverPdfEst = 1;
                }

                float e = curVertex.contri_float();

                if (e < 0.00001) 
                    break;
                /* lightVertexCount 在经过这个函数后会加 1 */
                pushVertexToLVC(curVertex, lightVertexCount, bufferBias); 
                CheckLightBufferState;
            }
            ray_direction = payload.ray_direction;
            ray_origin = payload.origin;
            if (payload.done || payload.depth > 50)
                break;
            payload.depth += 1;
        }
        lightPathCount++;
        if (lightPathCount >= lt_params.M_per_core)
            break;
        CheckLightBufferState; 
    }
    // printf("Light Trace %d get %d path and %d vertices\n", launch_index, lightPathCount, lightVertexCount);
    for (int i = lightVertexCount; i < lt_params.core_padding; i++)
    {
        lt_params.validState[lightVertexCount + bufferBias] = false;
        lightVertexCount++;
    } 
}

extern "C" __global__ void __miss__constant_radiance()
{
    Tracer::PayloadRadiance* prd = Tracer::getPRD();
    prd->done = true;
    prd->currentResult = make_float3(0);
    if (prd->depth == 0&&SKY.valid)
    {
        prd->result = prd->throughput* SKY.color(prd->ray_direction); 
    }
}


extern "C" __global__ void __miss__env__BDPTVertex()
{
    if (SKY.valid == false)
    {
        Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
        prd->done = true;
        return;
    }
    Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
    prd->done = true;

    if (prd->path.size == 1)
    {
        prd->result = SKY.color(prd->ray_direction);
        return;
    }
    //return;
    prd->path.push();
    BDPTVertex& MidVertex = prd->path.currentVertex(); 
    BDPTVertex& LastVertex = prd->path.lastVertex(); 
    MidVertex.normal = -prd->ray_direction;

    MidVertex.type = BDPTVertex::Type::ENV_MISS;
    MidVertex.uv = dir2uv(prd->ray_direction); 
    Tracer::lightSample light_sample;
    light_sample.ReverseSample(Tracer::params.lights[SKY.light_id], MidVertex.uv);

    float lightPdf = light_sample.pdf;

    float pdf_G = abs(dot(MidVertex.normal, prd->ray_direction) * dot(LastVertex.normal, prd->ray_direction));
    if (LastVertex.isOrigin)
    {
        MidVertex.flux = LastVertex.flux * pdf_G * SKY.color(prd->ray_direction);
    }
    else
    {
        MidVertex.flux = MidVertex.flux * LastVertex.flux * pdf_G * SKY.color(prd->ray_direction);
    }



    MidVertex.lastPosition = LastVertex.position;
    MidVertex.lastNormalProjection = abs(dot(LastVertex.normal, prd->ray_direction));

    //MidVertex.zoneId = SUBSPACE_NUM - lightMaterialId - 1;
    MidVertex.subspaceId = SKY.getLabel(prd->ray_direction);
    //MidVertex.zoneId = -1;
    MidVertex.lastZoneId = LastVertex.subspaceId;


    MidVertex.singlePdf = MidVertex.singlePdf;
    MidVertex.pdf = LastVertex.pdf * MidVertex.singlePdf;

    //MidVertex.dLast = LastVertex.d;

    MidVertex.depth = LastVertex.depth + 1;
     
      
    float3 dir = -MidVertex.normal;
    BDPTVertex virtual_light;
    rmis::construct_virtual_env_light(virtual_light, SKY.color(dir), light_sample.pdf, dir, SKY.getLabel(dir));
    float dd = rmis::light_hit_env(LastVertex, virtual_light);
    //printf("env hit compare %f %f\n", dd, 1.0 / MidVertex.d);
    MidVertex.RMIS_pointer = 1.0 / dd;

}
extern "C" __global__ void __miss__BDPTVertex()
{
    Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
    prd->done = true;
//    prd->result = SKY.color(prd->ray_direction);
}



RT_FUNCTION void PreTrace_buildPathInfo(BDPTVertex* eye, TrainData::nVertex_device light, preTracePath* path, preTraceConnection* conn, int pathSize)
{ 
    //check if the path is caustic path
    {
        path->is_caustic = false;
        BDPTVertex* eye_subpath_it = eye - eye->depth + 1;
        for (int i = 1; i < eye->depth; i++, eye_subpath_it++)
        {
            if (Shift::glossy(*eye_subpath_it))continue;
            else if (Shift::glossy(*(eye_subpath_it + 1)))
            {
                path->is_caustic = true;
                path->caustic_id = i - 1;
            }
            else
            {
                break;
            }
        }
    }

    path->valid = true;
    
    path->begin_ind = 0;
    path->end_ind = pathSize - 1;
    //path->end_ind = 0;
    //return;

    path->sample_pdf = 0;
     
    TrainData::nVertex_device n_eye = TrainData::nVertex_device(*eye, true);

    //printf("material ID %d %d %d\n", n_eye.materialId, eye->materialId, eye->depth);
    TrainData::nVertex_device n_next_eye = TrainData::nVertex_device(light, n_eye, true); 
    float3 seg_contri = n_eye.local_contri(light);

    path->sample_pdf = n_next_eye.pdf; 
    path->sample_pdf += n_eye.pdf * light.pdf;
    path->fix_pdf = n_next_eye.pdf;
    path->contri = eye->flux * light.forward_light(n_eye) * seg_contri;
    for (int i = 0; i < path->end_ind; i++)
    {
        conn[path->end_ind - i - 1] = preTraceConnection(n_eye, light);
        eye--;
        light = TrainData::nVertex_device(n_eye, light, false);
        n_eye = TrainData::nVertex_device(*eye, true);
    }
    float weight = (float3weight(path->contri) / path-> sample_pdf);
    if (isnan(weight))path->contri = make_float3(0);
    if (isinf(weight))path->contri = make_float3(0);
//    if (path->is_caustic == false) path->valid = false;// *= 100;
    //printf("pretrace path info%f %f %f\n", float3weight(path->contri), path->fix_pdf, weight);
}
RT_FUNCTION bool rr_acc_accept(int acc_num, unsigned int& seed)
{
    float r = rnd(seed); 
    if (1.0f / (acc_num + 1) > r)
    {
        return true;
    }
    return false;
}
#define PRETRACER_PADDING_VERTICES_CHECK(a) if (a >= pretracer_params.padding)break;
extern "C" __global__ void __raygen__TrainData()
{
    const uint3  launch_idx = optixGetLaunchIndex();
    const uint3  launch_dims = optixGetLaunchDimensions();
    const PreTraceParams& pretracer_params = Tracer::params.pre_tracer;
    const int    subframe_index = pretracer_params.iteration;
    unsigned int seed = tea<4>(launch_idx.y * launch_dims.x + launch_idx.x, subframe_index);


    const float3 eye = Tracer::params.eye;
    const float3 U = Tracer::params.U;
    const float3 V = Tracer::params.V;
    const float3 W = Tracer::params.W; 

    float3 normalizeV = normalize(V);

    const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

    const float2 d = 2.0f * make_float2(subpixel_jitter.x, subpixel_jitter.y) - 1.0f;

    float3 ray_direction = normalize(d.x * U + d.y * V + W);
    float3 ray_origin = eye;
    //printf("eye %f %f %f\n", eye.x, eye.y, eye.z);

    BDPTVertex buffer[PRETRACE_CONN_PADDING];
    int buffer_size = 0;

    int resample_number = 0;
     
    Tracer::PayloadBDPTVertex payload;
    payload.clear();
    payload.seed = seed;
    init_EyeSubpath(payload.path, ray_origin, ray_direction);

    int launch_index = launch_idx.x;
    unsigned int bufferBias = launch_index * pretracer_params.padding;
    preTracePath* currentPath = pretracer_params.paths + launch_index;
    preTraceConnection* currentConn = pretracer_params.conns + bufferBias;
    currentPath->valid = false;
    buffer[buffer_size] = payload.path.currentVertex();
    buffer_size++;

    while (true)
    { 
        int begin_depth = payload.path.size;
        Tracer::traceEyeSubPath(Tracer::params.handle, ray_origin, ray_direction,
            SCENE_EPSILON,  // tmin
            1e16f,  // tmax
            &payload); 
        if (payload.path.size == begin_depth)
        {
            break;
        } 
        if (payload.path.hit_lightSource())
        { 
            if (payload.path.currentVertex().type != BDPTVertex::Type::ENV_MISS && payload.path.size > 2 && rr_acc_accept(resample_number, payload.seed))
            {
                Tracer::lightSample light_sample;
                int light_id = payload.path.currentVertex().materialId;
                light_sample.ReverseSample(Tracer::params.lights[light_id], payload.path.currentVertex().uv);

                BDPTVertex light_vertex;
                init_vertex_from_lightSample(light_sample, light_vertex);
                PreTrace_buildPathInfo(buffer + buffer_size - 1, TrainData::nVertex_device(light_vertex ,false), currentPath, currentConn, buffer_size);
                resample_number++;
            }  
            break;
        } 
        buffer[buffer_size] = payload.path.currentVertex();
        buffer_size++;

        BDPTVertex& eye_subpath = payload.path.currentVertex();
        Tracer::lightSample light_sample;
        light_sample(payload.seed);
        float3 vis_vec = light_sample.position - eye_subpath.position;
        BDPTVertex light_vertex;
        init_vertex_from_lightSample(light_sample, light_vertex);
        if (Tracer::visibilityTest(Tracer::params.handle, eye_subpath, light_vertex)
            && rr_acc_accept(resample_number, payload.seed)) 
        {
            if ((light_vertex.is_DIRECTION() && dot(light_vertex.normal, eye_subpath.normal) < 0) ||
                (!light_vertex.is_DIRECTION() && dot(vis_vec, light_sample.normal()) < 0))
            {
                PreTrace_buildPathInfo(buffer + buffer_size - 1, TrainData::nVertex_device(light_vertex, false), currentPath, currentConn, buffer_size); 
                resample_number++; 
            }
             
        }
         
        if (payload.done || payload.depth > 50)
        {
            break;
        }
        PRETRACER_PADDING_VERTICES_CHECK(buffer_size);
        ray_direction = payload.ray_direction;
        ray_origin = payload.origin;
        payload.depth += 1;
    }

    int beginIndex = 0;
    if (currentPath->valid)
    {
        beginIndex += currentPath->end_ind - currentPath->begin_ind;
    }
    for (int i = beginIndex; i < pretracer_params.padding; i++)
    {
        currentConn[i].valid = false;        
    }
     
    currentPath->sample_pdf/= resample_number;
    currentPath->begin_ind += bufferBias;
    currentPath->end_ind += bufferBias; 
    currentPath->pixel_id = make_int2(Tracer::params.width * subpixel_jitter.x, Tracer::params.height * subpixel_jitter.y);
    if (currentPath->begin_ind == currentPath->end_ind && currentPath->valid == true)
    {
        currentPath->valid = false;
    }

    if (currentPath->is_caustic == false)
    {
        if (rnd(seed) > 1.0 / 4.0)currentPath->valid = false;
    }
}


RT_FUNCTION bool eye_step(Tracer::PayloadBDPTVertex& prd)
{
    float3 ray_origin = prd.origin;
    float3 ray_direction = prd.ray_direction;
    int origin_depth = prd.path.size;
    Tracer::traceEyeSubPath(Tracer::params.handle, ray_origin, ray_direction,
        SCENE_EPSILON,  // tmin
        1e16f,  // tmax
        &prd);


    prd.depth++;
    if (prd.path.size == origin_depth)
    {
        //miss
        prd.done = true;
        return false;
    }
    
#define ISLIGHTSOURCE(a) (a.type == BDPTVertex::Type::HIT_LIGHT_SOURCE||a.type == BDPTVertex::Type::ENV_MISS)
#define ISVALIDVERTEX(a) (fmaxf(a.flux / a.pdf)>= 0.00000001f)
    if (ISLIGHTSOURCE(prd.path.currentVertex()))
    {
        prd.done = true;
        return true;
    }

    if (!ISVALIDVERTEX(prd.path.currentVertex()))
    {
        prd.done = true;
        return false;
    }
    return true;

}
extern "C" __global__ void __raygen__TrainData_V2()
{
    const uint3  launch_idx = optixGetLaunchIndex();
    const uint3  launch_dims = optixGetLaunchDimensions();
    const PreTraceParams& pretracer_params = Tracer::params.pre_tracer;
    const int    subframe_index = pretracer_params.iteration;
    unsigned int seed = tea<16>(launch_idx.y * launch_dims.x + launch_idx.x, subframe_index);


    const float3 eye = Tracer::params.eye;
    const float3 U = Tracer::params.U;
    const float3 V = Tracer::params.V;
    const float3 W = Tracer::params.W;

    const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

    const float2 d = 2.0f * make_float2(subpixel_jitter.x, subpixel_jitter.y) - 1.0f;
    
    float3 ray_direction = normalize(d.x * U + d.y * V + W);
    float3 ray_origin = eye; 

    BDPTVertex buffer[PRETRACE_CONN_PADDING];
    int buffer_size = 0;

    int resample_number = 0;

    Tracer::PayloadBDPTVertex payload;
    payload.clear();
    payload.seed = seed;
    init_EyeSubpath(payload.path, ray_origin, ray_direction);

    int launch_index = launch_idx.x;
    unsigned int bufferBias = launch_index * pretracer_params.padding;
    preTracePath* currentPath = pretracer_params.paths + launch_index;
    preTraceConnection* currentConn = pretracer_params.conns + bufferBias;
    currentPath->valid = false;

    buffer[buffer_size] = payload.path.currentVertex();
    buffer_size++;
    while (true)
    {
        //bool hit_success = eye_step(payload);
        //if (!hit_success)break;

        //buffer[buffer_size] = payload.path.currentVertex();
        //buffer_size++;
        //PRETRACER_PADDING_VERTICES_CHECK(buffer_size);
        // if(payload.done == true)break;
        //if (!ISLIGHTSOURCE(payload.path.currentVertex()))
        //{
        //    continue;
        //}
        int begin_depth = payload.path.size;
        Tracer::traceEyeSubPath(Tracer::params.handle, ray_origin, ray_direction,
            SCENE_EPSILON,  // tmin
            1e16f,  // tmax
            &payload);
        if (payload.path.size == begin_depth)
        {
            break;
        }
        buffer[buffer_size] = payload.path.currentVertex();
        buffer_size++;
        //if(buffer_size == 1)
        //    printf("pos %f %f %f\n", payload.path.currentVertex().position.x, payload.path.currentVertex().position.y, payload.path.currentVertex().position.z);
        if (payload.path.hit_lightSource())
        { 
            if (payload.path.size > 2)
            {
                int light_id = payload.path.currentVertex().materialId;
                float sample_pdf = payload.path.currentVertex().pdf;
                TrainData::nVertex light_vertex;
                {
                    BDPTVertex eyeEndVertex = payload.path.currentVertex();

                    Tracer::lightSample light_sample;
                    light_sample.ReverseSample(Tracer::params.lights[light_id], payload.path.currentVertex().uv);

                    eyeEndVertex.normal = light_sample.normal();
                    eyeEndVertex.flux = light_sample.emission;
                    eyeEndVertex.pdf = light_sample.pdf; 
                    light_vertex = TrainData::nVertex_device(eyeEndVertex, false);
                    light_vertex.materialId = -1;
                }
                buffer_size--; 
                TrainData::pathInfo_sample& sample = *currentPath;

                sample.valid = true;
                sample.pixel_id = make_int2(Tracer::params.width * subpixel_jitter.x, Tracer::params.height * subpixel_jitter.y);

                TrainData::nVertex_device* light_nVertex_p = (TrainData::nVertex_device*)&light_vertex;
                TrainData::nVertex_device eye_nVertex = TrainData::nVertex_device(buffer[buffer_size - 1], true);
                float3 seg_contri = eye_nVertex.local_contri(*light_nVertex_p);
                sample.contri = buffer[buffer_size - 1].flux * light_nVertex_p->forward_light(eye_nVertex) * seg_contri;
                sample.choice_id = 0;

                if (buffer_size - 1 >= Tracer::params.pre_tracer.padding)
                {
                    sample.valid = false; 
                    break;
                }
                if (!(float3weight(sample.contri) > 0.0))
                {
                    sample.valid = false; 
                    break;
                }
                sample.sample_pdf = sample_pdf;
                sample.fix_pdf = sample_pdf;
                sample.begin_ind = bufferBias;
                int counter = 0;
                {
                    pretracer_params.conns[bufferBias + counter] = TrainData::pathInfo_node(eye_nVertex, *light_nVertex_p);
                    counter++;
                }
                light_nVertex_p = (TrainData::nVertex_device*)&light_vertex;

                eye_nVertex = TrainData::nVertex_device(buffer[buffer_size - 1], true);
                TrainData::nVertex_device light_nVertex; 
                  
                for (int i = 1; buffer_size - 1 - i > 0; i++)
                {
                    light_nVertex = TrainData::nVertex_device(eye_nVertex, *light_nVertex_p, false);

                    eye_nVertex = TrainData::nVertex_device(buffer[buffer_size - 1 - i], true);
                    light_nVertex_p = &light_nVertex;


                    pretracer_params.conns[bufferBias + counter] = TrainData::pathInfo_node(eye_nVertex, *light_nVertex_p);
                    counter++; 
                }
                sample.end_ind = sample.begin_ind + counter; 
                if (sample.fix_pdf < float3weight(sample.contri) * 0.00001)
                {
                    sample.fix_pdf = float3weight(sample.contri) * 0.00001;
                }  
            }
            break;
        }



        if (payload.done )
        {
            break;
        }
        PRETRACER_PADDING_VERTICES_CHECK(buffer_size);
        ray_direction = payload.ray_direction;
        ray_origin = payload.origin;
        payload.depth += 1;
    }

    int beginIndex = 0;
    if (currentPath->valid)
    {
        beginIndex += currentPath->end_ind - currentPath->begin_ind;
    }
    for (int i = beginIndex; i < pretracer_params.padding; i++)
    {
        currentConn[i].valid = false;
    }
     
    currentPath->pixel_id = make_int2(Tracer::params.width * subpixel_jitter.x, Tracer::params.height * subpixel_jitter.y);
    if (currentPath->begin_ind == currentPath->end_ind && currentPath->valid == true)
    { 
        currentPath->valid = false;
    }
}
