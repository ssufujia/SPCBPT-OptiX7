#include "hip/hip_runtime.h"

#include <optix.h>

#include <cuda/LocalGeometry.h>
#include <cuda/LocalShading.h>
#include <cuda/helpers.h>
#include <cuda/random.h>
#include <sutil/vec_math.h>
#include "BDPTVertex.h"
#include "cuProg.h"
#include "pathControl.h"
#include "rmis.h"

#define PT_BRDF_STRATEGY_ONLY
//#define PT_NEE_STRATEGY_ONLY

extern "C" __global__ void __anyhit__radiance()
{
    //optixIgnoreIntersection();
    return;
    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>(optixGetSbtDataPointer());
    if (hit_group_data->material_data.pbr.base_color_tex)
    {
        const LocalGeometry geom = getLocalGeometry(hit_group_data->geometry_data);
        const float         base_alpha = sampleTexture<float4>(hit_group_data->material_data.pbr.base_color_tex, geom).w;
        // force mask mode, even for blend mode, as we don't do recursive traversal.
        if (base_alpha < hit_group_data->material_data.alpha_cutoff)
            optixIgnoreIntersection();
    }
}

extern "C" __global__ void __anyhit__occlusion()
{
    Tracer::setPayloadOcclusion(0.f);
    return;
    optixTerminateRay();
    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>(optixGetSbtDataPointer());
    if (hit_group_data->material_data.pbr.base_color_tex)
    {
        const LocalGeometry geom = getLocalGeometry(hit_group_data->geometry_data);
        const float         base_alpha = sampleTexture<float4>(hit_group_data->material_data.pbr.base_color_tex, geom).w;

        if (hit_group_data->material_data.alpha_mode != MaterialData::ALPHA_MODE_OPAQUE)
        {
            if (hit_group_data->material_data.alpha_mode == MaterialData::ALPHA_MODE_MASK)
            {
                if (base_alpha < hit_group_data->material_data.alpha_cutoff)
                    optixIgnoreIntersection();
            }

            float attenuation = Tracer::getPayloadOcclusion() * (1.f - base_alpha);

            if (attenuation > 0.f)
            {
                Tracer::setPayloadOcclusion(attenuation);
                optixIgnoreIntersection();
            }
        }
    }
}

extern "C" __global__ void __closesthit__occlusion()
{
    Tracer::setPayloadOcclusion(0.f);
}
extern "C" __global__ void __closesthit__eyeSubpath_LightSource()
{
    Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>(optixGetSbtDataPointer());
    prd->done = true;
    const Light& light = Tracer::params.lights[hit_group_data->material_data.light_id];
    if (dot(prd->ray_direction, light.quad.normal) > 0)
    { 
        return;
    }

    const LocalGeometry          geom = getLocalGeometry(hit_group_data->geometry_data);
    float t_hit = optixGetRayTmax();
    float3 ray_direction = optixGetWorldRayDirection();
    float3 inver_ray_direction = -ray_direction;  
 

    prd->path.push();
    BDPTVertex& MidVertex = prd->path.currentVertex();// prd.stackP->v[(prd.stackP->size) % STACKSIZE];
    BDPTVertex& LastVertex = prd->path.lastVertex();// prd.stackP->v[(prd.stackP->size - 1) % STACKSIZE];
    
    MidVertex.position = geom.P;
    MidVertex.normal = light.quad.normal; 
    MidVertex.type = BDPTVertex::Type::HIT_LIGHT_SOURCE;
    MidVertex.uv = geom.texcoord->UV;
    Tracer::lightSample light_sample;
    light_sample.ReverseSample(light, MidVertex.uv);
    float lightPdf = light_sample.pdf;
     
    
    float pdf_G = abs(dot(MidVertex.normal, ray_direction) * dot(LastVertex.normal, ray_direction)) / (t_hit * t_hit);
    if (LastVertex.isOrigin)
    {
        MidVertex.flux = LastVertex.flux * pdf_G * light_sample.emission;
    }
    else
    {
        MidVertex.flux = MidVertex.flux * LastVertex.flux * pdf_G * light_sample.emission;
    }


    MidVertex.lastPosition = LastVertex.position;
    MidVertex.lastNormalProjection = abs(dot(LastVertex.normal, ray_direction));

    //MidVertex.zoneId = SUBSPACE_NUM - lightMaterialId - 1;
    MidVertex.subspaceId = light_sample.subspaceId;  
    MidVertex.lastZoneId = LastVertex.subspaceId;


    MidVertex.singlePdf = MidVertex.singlePdf * pdf_G / abs(dot(LastVertex.normal, ray_direction));
    MidVertex.pdf = LastVertex.pdf * MidVertex.singlePdf;

    //MidVertex.dLast = LastVertex.d;
    MidVertex.materialId = hit_group_data->material_data.light_id;

    MidVertex.depth = LastVertex.depth + 1;

    if (MidVertex.depth == 1)
    {
        MidVertex.RMIS_pointer = 1.0;         
        return;
    }

    BDPTVertex virtual_light;
    virtual_light.position = MidVertex.position;
    virtual_light.RMIS_pointer = 1;
    virtual_light.normal = MidVertex.normal;
    virtual_light.pdf = lightPdf;
    virtual_light.singlePdf = lightPdf;
    virtual_light.flux = light_sample.emission;
    virtual_light.subspaceId = MidVertex.subspaceId;
    virtual_light.isBrdf = false;
    //rtPrintf("%f %f\n", 1.0 / MidVertex.d, light_hit(LastVertex, virtual_light)); 
    MidVertex.RMIS_pointer = 1.0 / rmis::light_hit(LastVertex, virtual_light);   
}
extern "C" __global__ void __closesthit__lightsource()
{
    Tracer::PayloadRadiance* prd = Tracer::getPRD();

    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>(optixGetSbtDataPointer());
    /* 打到了哪个光源? */
    const Light& light = Tracer::params.lights[hit_group_data->material_data.light_id];
    const LocalGeometry          geom = getLocalGeometry(hit_group_data->geometry_data);
    /* 算一算打到点的采样 pdf（用于MIS） */
    Tracer::lightSample light_sample;
    light_sample.ReverseSample(light, geom.texcoord->UV);
    float t_hit = optixGetRayTmax();
    float3 ray_direction = optixGetWorldRayDirection();

    if ( /* 打中的光源法向要求与光线方向相反 */
        (dot(prd->ray_direction, light_sample.normal()) <= 0 ) && (
        /* 光源直击, L - E */
            (LE_ENABLE && prd->depth == 0) || (
            /* 是否是 S_ONLY */
                (!S_ONLY || prd->path_record) && (
                /* L - * - E，所有光路 */
                    LAE_ENABLE ||
                /* L - D - E */
                    (LDE_ENABLE && prd->depth == 1 && prd->path_record == 0b0) || 
                /* L - D - S - E */
                    (LDSE_ENABLE && prd->depth == 2 && prd->path_record == 0b01) ||
                /* L - D - S - D - E */
                    (LDSDE_ENABLE && prd->depth == 3 && prd->path_record == 0b010) ||
                /* L - S - * - E */
                    (LSAE_ENABLE && prd->depth > 0 && (prd->path_record & (1ll << (prd->depth-1)))) ||
                /* L - S - E */
                    (LSE_ENABLE && prd->depth == 1 && prd->path_record == 0b1) ||
                /* L - S - D - E */
                    (LSDE_ENABLE && prd->depth == 2 && prd->path_record == 0b10) 
                )
            )
        )
    )
    {
        /* PT 加 NEE 的 MIS */
        float MIS_weight = 1;
        if (prd->depth != 0 )
        {
            float pdf_hit = prd->pdf * abs(dot(ray_direction,light_sample.normal())) / (t_hit * t_hit);

            float pdf_area = light_sample.pdf;
            MIS_weight = pdf_hit / (pdf_area + pdf_hit);
        }
#ifdef PT_BRDF_STRATEGY_ONLY 
        MIS_weight = 1;
#endif 
#ifdef PT_NEE_STRATEGY_ONLY 
        MIS_weight = 0;
#endif 
        prd->result += prd->throughput * light_sample.emission * MIS_weight;
    }
    prd->done = true;
    return;
}

RT_FUNCTION void ColorTexSample(const LocalGeometry& geom, MaterialData::Pbr& pbr)
{
    float4 base_color = pbr.base_color;// *geom.color;
    if (pbr.base_color_tex)
    {
        const float4 base_color_tex = sampleTexture<float4>(pbr.base_color_tex, geom);

        // don't gamma correct the alpha channel.
        const float3 base_color_tex_linear = Tracer::linearize(make_float3(base_color_tex));
        //const float3 base_color_tex_linear = make_float3(base_color_tex);

        base_color = make_float4(base_color_tex_linear.x, base_color_tex_linear.y, base_color_tex_linear.z, base_color_tex.w);
    }
    pbr.base_color = base_color;

    return;
}
RT_FUNCTION void RoughnessAndMetallicTexSample(const LocalGeometry& geom, MaterialData::Pbr& pbr)
{
    //float  metallic  = hit_group_data->material_data.pbr.metallic;
    //float  roughness = hit_group_data->material_data.pbr.roughness;
    float4 mr_tex = make_float4(1.0f);
    if (pbr.metallic_roughness_tex) 
        mr_tex = sampleTexture<float4>(pbr.metallic_roughness_tex, geom);
    pbr.roughness *= mr_tex.y;
    pbr.metallic *= mr_tex.z;
    return;
}
RT_FUNCTION float3 NormalTexSample(const LocalGeometry& geom, const MaterialData& matData)
{
    //
    // compute direct lighting
    //

    float3 N = geom.N;
    if (matData.normal_tex)
    {
        const int texcoord_idx = matData.normal_tex.texcoord;
        const float4 NN =
            2.0f * sampleTexture<float4>(matData.normal_tex, geom) - make_float4(1.0f);

        // Transform normal from texture space to rotated UV space.
        const float2 rotation = matData.normal_tex.texcoord_rotation;
        const float2 NN_proj = make_float2(NN.x, NN.y);
        const float3 NN_trns = make_float3(
            dot(NN_proj, make_float2(rotation.y, -rotation.x)),
            dot(NN_proj, make_float2(rotation.x, rotation.y)),
            NN.z);

        N = normalize(NN_trns.x * normalize(geom.texcoord[texcoord_idx].dpdu) + NN_trns.y * normalize(geom.texcoord[texcoord_idx].dpdv) + NN_trns.z * geom.N);
    }

    // Flip normal to the side of the incomming ray
    if (dot(N, optixGetWorldRayDirection()) > 0.f)
        N = -N;
    return N;
}
extern "C" __global__ void __closesthit__lightSource_subpath()
{
    Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
    prd->done = true;

}

extern "C" __global__ void __closesthit__eyeSubpath()
{
    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>(optixGetSbtDataPointer());
    const LocalGeometry          geom = getLocalGeometry(hit_group_data->geometry_data);
    Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
    float t_hit = optixGetRayTmax();
    float3 ray_direction = optixGetWorldRayDirection();
    float3 inver_ray_direction = -ray_direction;
    MaterialData::Pbr currentPbr = hit_group_data->material_data.pbr;
    ColorTexSample(geom, currentPbr);
    RoughnessAndMetallicTexSample(geom, currentPbr);
    float3 N = geom.N;// NormalTexSample(geom, hit_group_data->material_data);
//    if (dot(N, ray_direction) > 0.f)
//        N = -N;
    prd->ray_direction = Tracer::Sample(currentPbr, N, inver_ray_direction, prd->seed);
    prd->pdf = Tracer::Pdf(currentPbr, N, inver_ray_direction, prd->ray_direction);
    prd->origin = geom.P;
    if (!(prd->pdf > 0.0f))
        prd->done = true;

    //    prd->path.size += 1;
    prd->path.push();
    BDPTVertex& MidVertex = prd->path.currentVertex();
    BDPTVertex& NextVertex = prd->path.nextVertex();
    BDPTVertex& LastVertex = prd->path.lastVertex();
    MidVertex.position = geom.P;
    MidVertex.normal = N;//这个在折射场景里需要进一步讨论
    MidVertex.type = BDPTVertex::Type::NORMALHIT;
    float pdf_G = abs(dot(MidVertex.normal, ray_direction) * dot(LastVertex.normal, ray_direction)) / (t_hit * t_hit);
 
    if (LastVertex.isOrigin)
    {
        MidVertex.flux = LastVertex.flux * pdf_G;
    }
    else
    {
        MidVertex.flux = MidVertex.flux * LastVertex.flux * pdf_G;
    }
    NextVertex.flux = Tracer::Eval(currentPbr, N, inver_ray_direction, prd->ray_direction) / (currentPbr.brdf ? abs(dot(MidVertex.normal, prd->ray_direction)) : 1.0f);
    NextVertex.singlePdf = prd->pdf;

    MidVertex.lastPosition = LastVertex.position;
    if (LastVertex.is_DIRECTION())
    {
        MidVertex.lastPosition = MidVertex.position - ray_direction;
    }

    MidVertex.color = make_float3(currentPbr.base_color);
    MidVertex.lastNormalProjection = abs(dot(LastVertex.normal, ray_direction));
    MidVertex.materialId = hit_group_data->material_data.id;

    labelUnit lu(MidVertex.position, MidVertex.normal, -ray_direction, false);
    MidVertex.subspaceId = lu.getLabel();
    MidVertex.lastZoneId = LastVertex.subspaceId;
    MidVertex.lastBrdf = LastVertex.isBrdf;
    MidVertex.isOrigin = false;
    MidVertex.depth = LastVertex.depth + 1;
    MidVertex.uv = geom.texcoord[0].UV;

    MidVertex.singlePdf = MidVertex.singlePdf * pdf_G / abs(dot(LastVertex.normal, ray_direction));
    MidVertex.pdf = LastVertex.pdf * MidVertex.singlePdf;
     
    //MidVertex.last_lum = Tracer::float3sum(LastVertex.flux / LastVertex.pdf);

    {
        MidVertex.lastSinglePdf = LastVertex.singlePdf;
        MidVertex.isLastVertex_direction = LastVertex.depth == 0 && (LastVertex.is_DIRECTION());
        if (MidVertex.depth == 1)
        {
            rmis::tracing_init_eye(MidVertex, LastVertex);
        }
        else
        {
            rmis::tracing_update_eye(MidVertex, LastVertex);
        }

        float r = rnd(prd->seed);

        float rr_rate = Tracer::rrRate(currentPbr);
        if (r > rr_rate)
        {
            prd->done = true;
        }
        else
        {
            NextVertex.singlePdf *= rr_rate;
            prd->throughput *= NextVertex.flux / prd->pdf / rr_rate * dot(N, prd->ray_direction);
        }
        return;
    }
}


extern "C" __global__ void __closesthit__eyeSubpath_LightSource_simple()
{
    Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>(optixGetSbtDataPointer());
    prd->done = true;
    const Light& light = Tracer::params.lights[hit_group_data->material_data.light_id];
    if (dot(prd->ray_direction, light.quad.normal) > 0)
    {
        return;
    }


    const LocalGeometry          geom = getLocalGeometry(hit_group_data->geometry_data);

    prd->path.push();
    BDPTVertex& MidVertex = prd->path.currentVertex();// prd.stackP->v[(prd.stackP->size) % STACKSIZE];
    BDPTVertex& LastVertex = prd->path.lastVertex();// prd.stackP->v[(prd.stackP->size - 1) % STACKSIZE];

    MidVertex.position = geom.P;
    MidVertex.normal = light.quad.normal;
    MidVertex.type = BDPTVertex::Type::HIT_LIGHT_SOURCE;
    MidVertex.uv = geom.texcoord->UV;


    MidVertex.materialId = hit_group_data->material_data.light_id;

    MidVertex.depth = LastVertex.depth + 1;

}
extern "C" __global__ void __closesthit__eyeSubpath_simple()
{ 
    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>(optixGetSbtDataPointer());
    const LocalGeometry          geom = getLocalGeometry(hit_group_data->geometry_data);
    Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
    float t_hit = optixGetRayTmax();
    float3 ray_direction = optixGetWorldRayDirection();
    float3 inver_ray_direction = -ray_direction;
    MaterialData::Pbr currentPbr = hit_group_data->material_data.pbr;
    ColorTexSample(geom, currentPbr);
    RoughnessAndMetallicTexSample(geom, currentPbr);
    float3 N = geom.N;// NormalTexSample(geom, hit_group_data->material_data);
//    if (dot(N, ray_direction) > 0.f)
//        N = -N;
    //prd->ray_direction = Tracer::Sample(currentPbr, N, inver_ray_direction, prd->seed); 
    prd->origin = geom.P;
    if (!(prd->pdf > 0.0f))
        prd->done = true;

    //    prd->path.size += 1;
    prd->path.push();
    BDPTVertex& MidVertex = prd->path.currentVertex();
    BDPTVertex& NextVertex = prd->path.nextVertex();
    BDPTVertex& LastVertex = prd->path.lastVertex();
    MidVertex.position = geom.P;
    MidVertex.normal = N;//这个在折射场景里需要进一步讨论
    MidVertex.type = BDPTVertex::Type::NORMALHIT; 
    MidVertex.color = make_float3(currentPbr.base_color); 

    MidVertex.materialId = hit_group_data->material_data.id;
      
    MidVertex.depth = LastVertex.depth + 1;
    MidVertex.uv = geom.texcoord[0].UV;
     
    //MidVertex.last_lum = Tracer::float3sum(LastVertex.flux / LastVertex.pdf);
     
}
extern "C" __global__ void __closesthit__lightSubpath()
{
    // printf("lightSubpath\n");
    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>(optixGetSbtDataPointer());
    const LocalGeometry          geom = getLocalGeometry(hit_group_data->geometry_data);
    Tracer::PayloadBDPTVertex* prd = Tracer::getPRD<Tracer::PayloadBDPTVertex>();
    float t_hit = optixGetRayTmax();
    float3 ray_direction = optixGetWorldRayDirection();
    float3 inver_ray_direction = -ray_direction;
    MaterialData::Pbr currentPbr = hit_group_data->material_data.pbr;
    ColorTexSample(geom, currentPbr);
    RoughnessAndMetallicTexSample(geom, currentPbr);
    float3 N = geom.N;
    // NormalTexSample(geom, hit_group_data->material_data);
    // if (dot(N, ray_direction) > 0.f)
    // N = -N;
    prd->ray_direction = Tracer::Sample(currentPbr, N, inver_ray_direction, prd->seed); 
    prd->pdf           = Tracer::Pdf(currentPbr, N, inver_ray_direction, prd->ray_direction);
    prd->origin        = geom.P;
    if (!(prd->pdf > 0.0f))
        prd->done = true;
    
//    prd->path.size += 1;
    prd->path.push();
    BDPTVertex& MidVertex = prd->path.currentVertex();
    BDPTVertex& NextVertex = prd->path.nextVertex();
    BDPTVertex& LastVertex = prd->path.lastVertex();
    MidVertex.position = geom.P;
    MidVertex.normal = N;//这个在折射场景里需要进一步讨论
    MidVertex.type = BDPTVertex::Type::NORMALHIT;
    float pdf_G = abs(dot(MidVertex.normal, ray_direction) * dot(LastVertex.normal, ray_direction)) / (t_hit * t_hit);

    if (LastVertex.is_DIRECTION())
        pdf_G = abs(dot(MidVertex.normal, ray_direction) * dot(LastVertex.normal, ray_direction));
    if (LastVertex.isOrigin)
        MidVertex.flux = LastVertex.flux * pdf_G;
    else
        MidVertex.flux = MidVertex.flux * LastVertex.flux * pdf_G;

    NextVertex.flux = Tracer::Eval(currentPbr, N, prd->ray_direction, -ray_direction) / (currentPbr.brdf ? abs(dot(MidVertex.normal, prd->ray_direction)) : 1.0f);

    NextVertex.singlePdf = prd->pdf;
     
    MidVertex.lastPosition = LastVertex.position;
    if (LastVertex.is_DIRECTION())
        MidVertex.lastPosition = MidVertex.position - ray_direction;

    MidVertex.color = make_float3(currentPbr.base_color);
    MidVertex.lastNormalProjection = abs(dot(LastVertex.normal, ray_direction));
    MidVertex.materialId = hit_group_data->material_data.id;

    labelUnit lu(MidVertex.position, MidVertex.normal, -ray_direction, true);
    MidVertex.subspaceId = lu.getLabel();
    MidVertex.lastZoneId = LastVertex.subspaceId;
    MidVertex.lastBrdf = LastVertex.isBrdf;
    MidVertex.isOrigin = false;
    MidVertex.depth = LastVertex.depth + 1;
    MidVertex.uv = geom.texcoord[0].UV;

    MidVertex.singlePdf = MidVertex.singlePdf * pdf_G / abs(dot(LastVertex.normal, ray_direction));
    MidVertex.pdf = LastVertex.pdf * MidVertex.singlePdf;

    MidVertex.last_lum = Tracer::float3sum(LastVertex.flux / LastVertex.pdf);

    MidVertex.lastSinglePdf = LastVertex.singlePdf;
    MidVertex.isLastVertex_direction = LastVertex.depth == 0 && (LastVertex.is_DIRECTION());
    if (LastVertex.isOrigin)
        rmis::tracing_init_light(MidVertex, LastVertex);
    else
        rmis::tracing_update_light(MidVertex, LastVertex);

    float r = rnd(prd->seed);
    float rr_rate = Tracer::rrRate(currentPbr);
    if (r > rr_rate)
        prd->done = true;
    else
        NextVertex.singlePdf *= rr_rate;
    return;
}
/* 这个函数应该是 PT 在打到普通面片时被调用 */
extern "C" __global__ void __closesthit__radiance() 
{
    // printf("__closesthit__radiance()\n");
    const Tracer::HitGroupData* hit_group_data = reinterpret_cast<Tracer::HitGroupData*>( optixGetSbtDataPointer() );
    const LocalGeometry          geom           = getLocalGeometry( hit_group_data->geometry_data );
    Tracer::PayloadRadiance* prd = Tracer::getPRD();

    /* Retrieve material data */
    MaterialData::Pbr currentPbr = hit_group_data->material_data.pbr;
    ColorTexSample(geom, currentPbr);
    RoughnessAndMetallicTexSample(geom, currentPbr);
    float3 N = geom.N;
    float3 in_dir = -prd->ray_direction;
    float3 result = make_float3( 0.0f );

    float rr_rate = Tracer::rrRate(currentPbr);
    prd->glossy_bounce = Shift::glossy(currentPbr) ? prd->glossy_bounce : false;


    /*  prd->path_record 用二进制按LSB到MSB的顺序编码了当前路径，0 代表 D, 1 代表 S */
    /* 比如 path_record 为 0010，depth 为 4，说明当前路径为 D - D - S - D - E */
    /* path_record 大小为 long long 以保证够用 */
    prd->path_record = (prd->path_record) | 
        ((long long) Shift::glossy(currentPbr) << prd->depth);
 
    /* 计算 NEE */
    int light_id = clamp(static_cast<int>(floorf(rnd(prd->seed) * Tracer::params.lights.count)), int(0), int(Tracer::params.lights.count - 1));
    Light light = Tracer::params.lights[light_id];
    if (light.type == Light::Type::QUAD)
    {
        Tracer::lightSample light_sample;
        light_sample(light, prd->seed);

        // TODO: optimize
        const float  L_dist = length(light_sample.position - geom.P);
        const float3 L = (light_sample.position - geom.P) / L_dist;
        const float3 V = -normalize(optixGetWorldRayDirection());
        const float3 H = normalize(L + V);
        const float3 LN = light.quad.normal;
        const float  L_dot_LN = dot(-L, LN);
        const float  N_dot_L = abs(dot(N, L));
        const float  N_dot_V = abs(dot(N, V));
        const float  N_dot_H  = dot(N, H);
        const float  V_dot_H  = dot(V, H);
        if (N_dot_L > 0.0f && N_dot_V > 0.0f && L_dot_LN > 0.0f)
        {
            const float tmin = 0.001f;           // TODO
            const float tmax = L_dist - 0.001f;  // TODO
            const float attenuation = 1;//// Tracer::traceOcclusion(Tracer::params.handle, geom.P, L, tmin, tmax);
            if (attenuation > 0.f)
            {
                prd->vis_pos_A = geom.P;
                prd->vis_pos_B = light_sample.position;
                float3 eval = Tracer::Eval(currentPbr, N, L, V);

                float MIS_weight = 1;
                {
                    float pdf_area = light_sample.pdf;
                    float pdf_hit = Tracer::Pdf(currentPbr, N, V, L, geom.P, true) * abs(L_dot_LN) / (L_dist * L_dist) * rr_rate;
                    MIS_weight = pdf_area / (pdf_hit + pdf_area);
                }
#ifdef PT_BRDF_STRATEGY_ONLY 
                MIS_weight = 0;
#endif // PT_BRDF_STRATEGY_ONLY 

#ifdef PT_NEE_STRATEGY_ONLY 
                MIS_weight = 1;
#endif // PT_BRDF_STRATEGY_ONLY 
                result += prd->throughput * light_sample.emission * attenuation / light_sample.pdf 
                    * N_dot_L * L_dot_LN / L_dist / L_dist * eval * MIS_weight ;// *make_float3(1.0, 0.0, 1.0);
            }
        }
    }
    else if (light.type == Light::Type::ENV)
    {
        Tracer::lightSample light_sample;
        light_sample(light, prd->seed);

        const float3 V = -normalize(optixGetWorldRayDirection());
        const float3 L = light_sample.direction;
        float L_dot_N = dot(light_sample.direction, N);
        if ( L_dot_N > 0.0)
        {
            prd->vis_pos_A = geom.P;
            prd->vis_pos_B = geom.P + light_sample.direction * SKY.r * 10;
//            printf("light_sample dir %f %f %f\n", light_sample.direction.x, light_sample.direction.y, light_sample.direction.z);
            float3 eval = Tracer::Eval(currentPbr, N, L, V);
            result += prd->throughput * light_sample.emission / light_sample.pdf * eval * L_dot_N;
        }
    }

    //const LocalGeometry geom = getLocalGeometry(hit_group_data->geometry_data);
    //result = make_float3(geom.texcoord[1].UV.x, geom.texcoord[0].UV.y ,0.0) + make_float3(1.0,1.0,0);
    //result = make_float3(currentPbr.base_color);  
     
    //prd->done = true;
    //prd->depth += 1;  
    //prd->result += result;


    //if (prd->depth > 5) result *= 0;

    prd->currentResult += result;
    prd->origin = geom.P;

    if (rnd(prd->seed) > rr_rate)
    {
        prd->done = true;
    }
    else
    {
        prd->ray_direction = Tracer::Sample(currentPbr, N, in_dir, prd->seed, geom.P, true); 
        float pdf = Tracer::Pdf(currentPbr, N, in_dir, prd->ray_direction, geom.P, true);

        if (isRefract(N, in_dir, prd->ray_direction) && prd->depth == 1 && dot(in_dir, N) > 0 && Shift::glossy(currentPbr))
        {
            float3 bsdf = Tracer::Eval(currentPbr, N, in_dir, prd->ray_direction);
            float cos_in = abs(dot(in_dir, N));
            float cos_out = abs(dot(prd->ray_direction, N));
            float sin_in = sqrt(1 - cos_in * cos_in);
            float sin_out = sqrt(1 - cos_out * cos_out);
        }
        if (pdf > 0.0f)
        {
            prd->throughput *= Tracer::Eval(currentPbr, N, in_dir, prd->ray_direction) * abs(dot(prd->ray_direction, N)) / pdf / rr_rate;
            prd->pdf = pdf * rr_rate;
        }
        else
        {
            prd->done = true;
        }
    }
     
    //Tracer::setPayloadResult( result );
}
